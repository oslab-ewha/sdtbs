#include "hip/hip_runtime.h"
#include "sdtbs_cu.h"

#include <pthread.h>
#include <unistd.h>

__device__ tbs_type_t	d_tbs_type;
__device__ skrun_t	*d_skruns;
__device__ unsigned	*d_mtbs_done_cnts;

static skrun_t	*g_skruns;
static unsigned	*g_mtbs_done_cnts;

static unsigned	*info_n_mtbs;

static BOOL	*skrun_dones;
static unsigned	skrid_done_min;
static unsigned	cur_skrid_host;

static BOOL	checker_done;
static pthread_t	checker;
static pthread_mutex_t	mutex = PTHREAD_MUTEX_INITIALIZER;
static pthread_cond_t	cond = PTHREAD_COND_INITIALIZER;

static hipStream_t	strm_submit;

#define SK_PROTO(name)	__device__ int name(void *args[])
#define SK_FUNCS(base)	SK_PROTO(base);

SK_FUNCS(loopcalc)
SK_FUNCS(mklc)
SK_FUNCS(gma)
SK_FUNCS(lma)
SK_FUNCS(kmeans)

static __device__ int
run_sub_kernel_func(skid_t skid, void *args[])
{
	switch (skid) {
	case LOOPCALC:
		return loopcalc(args);
	case MKLC:
		return mklc(args);		
	case GMA:
		return gma(args);
	case LMA:
		return lma(args);
	case KMEANS:
		return kmeans(args);
	default:
		return 0;
	}
}

__device__ void
run_sub_kernel(skrid_t skrid)
{
	skrun_t	*skr;
	int	res;

	skr = &d_skruns[skrid - 1];
	res = run_sub_kernel_func(skr->skid, skr->args);
	if (get_threadIdxX() == 0)
		skr->res = res;
}

__global__ void
sub_kernel_func(skrid_t skrid)
{
	run_sub_kernel(skrid);
}

static skrid_t
submit_skrun(skid_t skid, dim3 dimGrid, dim3 dimBlock, void *args[])
{
	skrid_t	skrid;
	skrun_t	skrun;

	skrun.skid = skid;
	skrun.dimGrid = dimGrid;
	skrun.dimBlock = dimBlock;
	memcpy(skrun.args, args, sizeof(void *) * MAX_ARGS);
	skrun.res = 0;
	skrun.n_tbs = dimGrid.x * dimGrid.y;
	skrun.n_mtbs_per_tb = dimBlock.x * dimBlock.y / N_THREADS_PER_mTB;

	pthread_mutex_lock(&mutex);

	skrid = cur_skrid_host + 1;
	info_n_mtbs[skrid - 1] = skrun.n_tbs * skrun.n_mtbs_per_tb;

	hipMemcpyAsync(g_skruns + cur_skrid_host, &skrun, sizeof(skrun_t), hipMemcpyHostToDevice, strm_submit);
	hipStreamSynchronize(strm_submit);

	cur_skrid_host++;

	pthread_mutex_unlock(&mutex);

	return skrid;
}

skrid_t
launch_kernel(skid_t skid, hipStream_t strm, dim3 dimGrid, dim3 dimBlock, void *args[])
{
	skrid_t	skrid;

	skrid = submit_skrun(skid, dimGrid, dimBlock, args);

	if (sched->type == TBS_TYPE_HW)
		sub_kernel_func<<<dimGrid, dimBlock, 0, strm>>>(skrid);
	return skrid;
}

static void
wait_skrun(skrid_t skrid)
{
	pthread_mutex_lock(&mutex);

	while (!checker_done && !skrun_dones[skrid - 1])
		pthread_cond_wait(&cond, &mutex);

	pthread_mutex_unlock(&mutex);
}

void
wait_kernel(skrid_t skrid, hipStream_t strm, int *pres)
{
	skrun_t	*skr;
	int	res;

	if (sched->type == TBS_TYPE_HW)
		hipStreamSynchronize(strm);
	else
		wait_skrun(skrid);

	skr = g_skruns + (skrid - 1);
	hipMemcpyAsync(&res, &skr->res, sizeof(int), hipMemcpyDeviceToHost, strm);
	hipStreamSynchronize(strm);
	*pres = res;
}

static void
notify_done_skruns(unsigned *mtbs_done_cnts, unsigned n_checks)
{
	unsigned	min_new = skrid_done_min;
	BOOL		notify = FALSE;
	unsigned	i;

	pthread_mutex_lock(&mutex);

	for (i = 0; i < n_checks; i++) {
		if (skrun_dones[i + skrid_done_min])
			continue;
		if (mtbs_done_cnts[i] == info_n_mtbs[i + skrid_done_min]) {
			notify = TRUE;
			skrun_dones[i + skrid_done_min] = TRUE;
			if (min_new == i + skrid_done_min) {
				min_new++;
			}
		}
	}
	skrid_done_min = min_new;
	if (notify)
		pthread_cond_broadcast(&cond);
	pthread_mutex_unlock(&mutex);
}

static void *
skruns_checkfunc(void *arg)
{
	hipStream_t	strm;

	hipStreamCreate(&strm);

	while (!checker_done) {
		unsigned	n_checks = cur_skrid_host - skrid_done_min;
		if (n_checks > 0) {
			unsigned	*mtbs_done_cnts = (unsigned *)malloc(sizeof(unsigned) * n_checks);
			hipMemcpyAsync(mtbs_done_cnts, g_mtbs_done_cnts + skrid_done_min, sizeof(unsigned) * n_checks, hipMemcpyDeviceToHost, strm);
			hipStreamSynchronize(strm);
			notify_done_skruns(mtbs_done_cnts, n_checks);
			free(mtbs_done_cnts);
		}
		usleep(100);
	}

	hipStreamDestroy(strm);
	return NULL;
}

__global__ void
kernel_init_skrun(tbs_type_t type, skrun_t *skruns, unsigned *mtbs_done_cnts)
{
	int	i;

	d_tbs_type = type;
	d_skruns = skruns;
	d_mtbs_done_cnts = mtbs_done_cnts;
	for (i = 0; i < MAX_QUEUED_KERNELS; i++) {
		skruns[i].skid = 0;
		mtbs_done_cnts[i] = 0;
	}
}

void
init_skrun(void)
{
	hipError_t	err;

	hipStreamCreate(&strm_submit);

	hipMalloc(&g_skruns, sizeof(skrun_t) * MAX_QUEUED_KERNELS);
	hipMalloc(&g_mtbs_done_cnts, sizeof(unsigned) * MAX_QUEUED_KERNELS);

	info_n_mtbs = (unsigned *)calloc(MAX_QUEUED_KERNELS, sizeof(unsigned));
	skrun_dones = (BOOL *)calloc(MAX_QUEUED_KERNELS, sizeof(BOOL));

	pthread_create(&checker, NULL, skruns_checkfunc, NULL);

	dim3 dimGrid(1,1), dimBlock(1,1);
	kernel_init_skrun<<<dimGrid, dimBlock>>>(sched->type, g_skruns, g_mtbs_done_cnts);
	err = hipGetLastError();
	if (err != hipSuccess)
		error("failed to initialize skrun: %s\n", hipGetErrorString(err));
	else
		hipDeviceSynchronize();
}

void
fini_skrun(void)
{
	void	*retval;

	checker_done = TRUE;
	pthread_join(checker, &retval);
}
