#include "hip/hip_runtime.h"
#include "sdtbs_cu.h"

extern __device__ void setup_dyn_sched(fedkern_info_t *fkinfo);
extern __device__ unsigned get_brid_dyn(BOOL *pis_primary_mtb);

__device__ int loopcalc(void *args[]);
__device__ int gma(void *args[]);

__device__ static int
run_bench(int skid, void *args[])
{
	switch (skid) {
	case 1:
		return loopcalc(args);
	case 2:
		return gma(args);
	default:
		return 0;
	}
}

static __device__ void
kernel_macro_TB_static_sched(fedkern_info_t *fkinfo)
{
	benchrun_k_t	*brk;
	unsigned	brid;
	int	n_mtbs_per_width;
	int	idx;
	int	res;

	n_mtbs_per_width = blockDim.x / N_THREADS_PER_mTB;

	idx = get_smid() * fkinfo->n_max_mtbs_per_sm + n_mtbs_per_width * threadIdx.y + threadIdx.x / N_THREADS_PER_mTB;

	brid = fkinfo->brids[idx];
	if (brid == 0)
		return;
	brk = &fkinfo->bruns[brid - 1];

	res = run_bench(brk->skid, brk->args);

	if (brk->primary_mtb_idx == idx + 1 && threadIdx.x % N_THREADS_PER_mTB == 0) {
		brk->res = res;
	}
}

static __device__ void
kernel_macro_TB_dynamic_sched(fedkern_info_t *fkinfo)
{
	if (threadIdx.x == 0 && threadIdx.y == 0) {
		setup_dyn_sched(fkinfo);
	}
	__syncthreads();

	while (TRUE) {
		benchrun_k_t	*brk;
		unsigned	brid;
		int	res;
		BOOL	is_primary_mtb;

		brid = get_brid_dyn(&is_primary_mtb);
		if (brid == 0)
			return;
		brk = &fkinfo->bruns[brid - 1];
		res = run_bench(brk->skid, brk->args);

		if (is_primary_mtb)
			brk->res = res;
	}
}

__global__ static void
kernel_macro_TB(fedkern_info_t *fkinfo)
{
	if (fkinfo->sched_id == 0)
		kernel_macro_TB_static_sched(fkinfo);
	else
		kernel_macro_TB_dynamic_sched(fkinfo);
}

static BOOL
launch_macro_TB(fedkern_info_t *fkinfo)
{
	hipError_t	err;

	dim3 dimGrid(n_sm_count, 1);
	dim3 dimBlock(n_threads_per_MTB, 1);

	kernel_macro_TB<<<dimGrid, dimBlock, 0>>>(fkinfo);

	err = hipGetLastError();
	if (err != hipSuccess) {
		error("kernel launch error: %s\n", hipGetErrorString(err));
		return FALSE;
	}

	hipDeviceSynchronize();
	return TRUE;
}

static void
collect_results(fedkern_info_t *fkinfo)
{
	benchrun_t	*brun;
	int	i;

	for (i = 0, brun = benchruns; i < n_benches; i++, brun++) {
		brun->res = fkinfo->bruns[i].res;
	}
}

extern "C" BOOL
run_sd_tbs(unsigned *pticks)
{
	fedkern_info_t	*fkinfo;
	fedkern_info_t	*d_fkinfo;

	if (!setup_gpu_devinfo()) {
		error("no gpu found");
		return FALSE;
	}

	fkinfo = setup_fedkern_info();

	hipMalloc(&d_fkinfo, fkinfo->size);

	run_schedule(fkinfo);

	hipMemcpy(d_fkinfo, fkinfo, fkinfo->size, hipMemcpyHostToDevice);

	init_tickcount();

	if (!launch_macro_TB(d_fkinfo))
		return FALSE;

	*pticks = get_tickcount();

	hipMemcpy(fkinfo, d_fkinfo, fkinfo->size, hipMemcpyDeviceToHost);
	collect_results(fkinfo);

	hipFree(d_fkinfo);

	return TRUE;
}
