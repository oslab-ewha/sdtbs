#include "hip/hip_runtime.h"
#include "sdtbs_cu.h"

__device__ int loopcalc(void *args[]);
__device__ int gma(void *args[]);

__device__ static uint
get_smid(void)
{
	uint	ret;
	asm("mov.u32 %0, %smid;" : "=r"(ret));
	return ret;
}

__global__ static void
kernel_macro_TB(int n_mtbs_per_sm, micro_tb_t *mtbs)
{
	micro_tb_t	*mtb;
	int	n_mtbs_per_width;
	int	res;

	n_mtbs_per_width = blockDim.x / N_THREADS_PER_mTB;
	mtb = mtbs + get_smid() * n_mtbs_per_sm + n_mtbs_per_width * threadIdx.y + threadIdx.x / N_THREADS_PER_mTB;

	switch (mtb->skid) {
	case 1:
		res = loopcalc(mtb->args);
		break;
	case 2:
		res = gma(mtb->args);
		break;
	default:
		goto out;
	}
	if (threadIdx.x % 32 == 0)
		mtb->args[0] = (void *)(long long)res;
out:
	__syncthreads();
}

static void
launch_macro_TB(int n_mtbs_per_sm, micro_tb_t *mtbs)
{
	hipError_t	err;

	dim3 dimGrid(n_sm_count, 1);
	dim3 dimBlock(n_threads_per_tb, 1);

	kernel_macro_TB<<<dimGrid, dimBlock, 0>>>(n_mtbs_per_sm, mtbs);

	err = hipGetLastError();
	if (err != hipSuccess) {
		error("kernel launch error: %s\n", hipGetErrorString(err));
		return;
	}

	hipDeviceSynchronize();
}

extern "C" void
run_sd_tbs(void)
{
	micro_tb_t	*d_mtbs;

	setup_gpu_devinfo();
	setup_micro_tbs();

	hipMalloc(&d_mtbs, n_mtbs * sizeof(micro_tb_t));

	run_schedule();

	hipMemcpy(d_mtbs, mtbs, n_mtbs * sizeof(micro_tb_t), hipMemcpyHostToDevice);

	launch_macro_TB(n_mtbs_per_sm, d_mtbs);

	hipFree(d_mtbs);
}
