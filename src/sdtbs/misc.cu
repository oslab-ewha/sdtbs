#include "sdtbs.h"

extern "C" BOOL
select_gpu_device(unsigned devno)
{
	if (hipSetDevice(devno) != 0)
		return FALSE;
	return TRUE;
}

