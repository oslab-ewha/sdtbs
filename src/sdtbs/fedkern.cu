#include "sdtbs_cu.h"

extern unsigned	n_max_mtbs_per_sm;

extern void setup_fedkern_info_static(fedkern_info_t *fkinfo);
extern void setup_fedkern_info_dyn(fedkern_info_t *fkinfo);
extern void setup_fedkern_info_kernel_static(fedkern_info_t *fkinfo);
extern void setup_fedkern_info_kernel_dyn(fedkern_info_t *fkinfo);
extern void free_fedkern_info_static(fedkern_info_t *fkinfo);
extern void free_fedkern_info_dyn(fedkern_info_t *fkinfo);

fedkern_info_t *
create_fedkern_info(void)
{
	fedkern_info_t	*fkinfo;

	fkinfo = (fedkern_info_t *)calloc(1, sizeof(fedkern_info_t));

	fkinfo->n_sm_count = n_sm_count;
	fkinfo->sched_id = sched->use_static_sched ? 0: sched_id;
	fkinfo->n_mtbs = n_mtbs_submitted;
	fkinfo->n_max_mtbs_per_sm = n_max_mtbs_per_sm;
	fkinfo->n_max_mtbs_per_MTB = n_max_mtbs_per_sm / n_MTBs_per_sm;
	fkinfo->n_tbs = n_tbs_submitted;

	if (sched->use_static_sched)
		setup_fedkern_info_static(fkinfo);
	else
		setup_fedkern_info_dyn(fkinfo);

	return fkinfo;
}

fedkern_info_t *
create_fedkern_info_kernel(fedkern_info_t *fkinfo)
{
	fedkern_info_t	*d_fkinfo;

	if (sched->use_static_sched)
		setup_fedkern_info_kernel_static(fkinfo);
	else
		setup_fedkern_info_kernel_dyn(fkinfo);

	hipMalloc(&d_fkinfo, sizeof(fedkern_info_t));
	hipMemcpy(d_fkinfo, fkinfo, sizeof(fedkern_info_t), hipMemcpyHostToDevice);

	return d_fkinfo;
}

void
free_fedkern_info(fedkern_info_t *fkinfo)
{
	if (sched->use_static_sched)
		free_fedkern_info_static(fkinfo);
	else
		free_fedkern_info_dyn(fkinfo);	
	free(fkinfo);
}

void
assign_fedkern_brun(fedkern_info_t *fkinfo,  benchrun_t *brun, unsigned char brid)
{
	benchrun_k_t    *brk;

	brk = &fkinfo->bruns[brid - 1];
	brk->skid = brun->info->skid;
	memcpy(brk->args, brun->args, sizeof(void *) * MAX_ARGS);
	brk->dimGrid = brun->dimGrid;
	brk->dimBlock = brun->dimBlock;
	brk->n_mtbs_per_tb = brun->dimBlock.x * brun->dimBlock.y / N_THREADS_PER_mTB;
}

void
wait_fedkern_initialized(fedkern_info_t *d_fkinfo)
{
	hipStream_t	strm;

	hipStreamCreate(&strm);

	while (TRUE) {
		BOOL	initialized = FALSE;

		hipMemcpyAsync(&initialized, &d_fkinfo->initialized, sizeof(BOOL), hipMemcpyDeviceToHost, strm);
		hipStreamSynchronize(strm);
		if (initialized)
			break;
	}
	hipStreamDestroy(strm);
}
