#include "sdtbs_cu.h"

#define SIZE_ARGS	(sizeof(void *) * MAX_ARGS)

static hipStream_t	strms[MAX_BENCHES];

extern "C" BOOL
run_native_tbs(unsigned *pticks)
{
	benchrun_t	*brun;
	void 	**d_args_brun;
	int	i;

	hipMalloc(&d_args_brun, SIZE_ARGS * n_benches);
	for (i = 0, brun = benchruns; i < n_benches; i++, brun++) {
		hipStreamCreate(strms + i);
		hipMemcpy((char *)d_args_brun + SIZE_ARGS * i, brun->args, SIZE_ARGS, hipMemcpyHostToDevice);
	}

	init_tickcount();

	for (i = 0, brun = benchruns; i < n_benches; i++, brun++) {
		int	ret;

		ret = brun->info->bench_func(strms[i], brun->n_grid_width, brun->n_grid_height, brun->n_tb_width,
					     brun->n_tb_height, (void **)((char *)d_args_brun + SIZE_ARGS * i));
		if (ret < 0)
			return FALSE;
	}

	for (i = 0; i < n_benches; i++)
		hipStreamSynchronize(strms[i]);

	*pticks = get_tickcount();

	for (i = 0, brun = benchruns; i < n_benches; i++, brun++) {
		hipMemcpy(brun->args, d_args_brun + SIZE_ARGS * i, SIZE_ARGS, hipMemcpyDeviceToHost);
		brun->res = (int)(long long)brun->args[0];
	}
	hipFree(d_args_brun);

	return TRUE;
}
