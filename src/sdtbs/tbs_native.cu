#include "sdtbs_cu.h"

extern "C" void
run_native_tbs(void)
{
	benchrun_t	*brun;
	hipStream_t	strm;
	int	i;

	brun = benchruns;
	hipStreamCreate(&strm);
	for (i = 0; i < n_benches; i++, brun++) {
		void 	**d_args;

		hipMalloc(&d_args, sizeof(void *) * MAX_ARGS);
		hipMemcpy(d_args, brun->args, sizeof(void *) * MAX_ARGS, hipMemcpyHostToDevice);
		brun->info->bench_func(strm, brun->n_tbs_x, brun->n_tbs_y, brun->n_threads_x, brun->n_threads_y, d_args);
		hipStreamSynchronize(strm);
		hipFree(d_args);
	}
}
