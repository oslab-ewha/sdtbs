#include "sdtbs_cu.h"

/* use MAX_ARGS + 1 for result value */
#define SIZE_ARGS	(sizeof(void *) * MAX_ARGS)

static hipStream_t	strms[MAX_BENCHES];

BOOL
run_native_tbs(unsigned *pticks)
{
	benchrun_t	*brun;
	void 	**d_args_brun;
	int	*d_benches_res;
	int	i;

	hipMalloc(&d_benches_res, sizeof(int) * n_benches);
	hipMalloc(&d_args_brun, SIZE_ARGS * n_benches);
	for (i = 0, brun = benchruns; i < n_benches; i++, brun++) {
		hipStreamCreate(strms + i);
		hipMemcpy((char *)d_args_brun + SIZE_ARGS * i, brun->args, SIZE_ARGS, hipMemcpyHostToDevice);
	}

	for (i = 0, brun = benchruns; i < n_benches; i++, brun++) {
		bench_func_t	bench;
		hipError_t	err;

		bench = sched->type == TBS_TYPE_HW_RELOC ? brun->info->bench_func: brun->info->bench_func_noreloc;
		bench(strms[i], brun->dimGrid, brun->dimBlock, (void **)((char *)d_args_brun + SIZE_ARGS * i), d_benches_res + i);
		err = hipGetLastError();
		if (err != hipSuccess) {
			printf("error: %s\n", hipGetErrorString(err));
			return FALSE;
		}
	}

	init_tickcount();

	for (i = 0; i < n_benches; i++)
		hipStreamSynchronize(strms[i]);

	*pticks = get_tickcount();

	hipFree(d_args_brun);

	for (i = 0, brun = benchruns; i < n_benches; i++, brun++) {
		hipMemcpy(&brun->res, d_benches_res + i, sizeof(int), hipMemcpyDeviceToHost);
	}
	hipFree(d_benches_res);

	return TRUE;
}
