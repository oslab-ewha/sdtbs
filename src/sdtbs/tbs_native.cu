#include "sdtbs_cu.h"

extern "C" BOOL
run_native_tbs(void)
{
	benchrun_t	*brun;
	hipStream_t	strm;
	int	i;

	brun = benchruns;
	hipStreamCreate(&strm);
	for (i = 0; i < n_benches; i++, brun++) {
		void 	**d_args;
		int	ret;

		hipMalloc(&d_args, sizeof(void *) * MAX_ARGS);
		hipMemcpy(d_args, brun->args, sizeof(void *) * MAX_ARGS, hipMemcpyHostToDevice);
		ret = brun->info->bench_func(strm, brun->n_grid_width, brun->n_grid_height, brun->n_tb_width, brun->n_tb_height, d_args);
		if (ret < 0) {
			return FALSE;
		}
		hipStreamSynchronize(strm);
		hipMemcpy(brun->args, d_args, sizeof(void *) * MAX_ARGS, hipMemcpyDeviceToHost);
		hipFree(d_args);
		brun->res = (int)(long long)brun->args[0];
	}

	return TRUE;
}
