#include "sdtbs_cu.h"

extern "C" void
run_native_tbs(void)
{
	benchrun_t	*brun;
	hipStream_t	strm;
	int	i;

	brun = benchruns;
	hipStreamCreate(&strm);
	for (i = 0; i < n_benches; i++, brun++) {
		void 	**d_args;

		hipMalloc(&d_args, sizeof(void *) * MAX_ARGS);
		hipMemcpy(d_args, brun->args, sizeof(void *) * MAX_ARGS, hipMemcpyHostToDevice);
		brun->info->bench_func(strm, brun->n_grid_width, brun->n_grid_height, brun->n_tb_width, brun->n_tb_height, d_args);
		hipStreamSynchronize(strm);
		hipFree(d_args);
	}
}
