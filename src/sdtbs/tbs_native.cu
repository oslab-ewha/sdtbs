#include "sdtbs.h"

#include "benchmgr.h"

extern "C" void
run_native_tbs(void)
{
	benchrun_t	*brun;
	hipStream_t	strm;
	int	i;

	brun = benchruns;
	hipStreamCreate(&strm);
	for (i = 0; i < n_benches; i++, brun++) {
		int	*d_args;

		hipMalloc(&d_args, sizeof(int) * MAX_ARGS);
		hipMemcpy(d_args, brun->args, sizeof(int) * MAX_ARGS, hipMemcpyHostToDevice);
		brun->info->bench_native(strm, brun->n_tbs_x, brun->n_tbs_y, brun->n_threads_x, brun->n_threads_y, d_args);
		hipStreamSynchronize(strm);
		hipFree(d_args);
	}
}
