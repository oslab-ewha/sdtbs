#include "sdtbs_cu.h"

/* use MAX_ARGS + 1 for result value */
#define SIZE_ARGS	(sizeof(void *) * MAX_ARGS)

static hipStream_t	strms[MAX_BENCHES];

BOOL
run_native_tbs(unsigned *pticks)
{
	benchrun_t	*brun;
	void 	**d_args_brun;
	int	*d_benches_res;
	int	i;

	hipMalloc(&d_benches_res, sizeof(int) * n_benches);
	hipMalloc(&d_args_brun, SIZE_ARGS * n_benches);
	for (i = 0, brun = benchruns; i < n_benches; i++, brun++) {
		hipStreamCreate(strms + i);
		hipMemcpy((char *)d_args_brun + SIZE_ARGS * i, brun->args, SIZE_ARGS, hipMemcpyHostToDevice);
	}

	init_tickcount();

	for (i = 0, brun = benchruns; i < n_benches; i++, brun++) {
		dim3	dimGrid(brun->n_grid_width, brun->n_grid_height);
		dim3	dimBlock(brun->n_tb_width, brun->n_tb_height);
		bench_func_t	bench;
		hipError_t	err;

		bench = sched->use_relocatable ? brun->info->bench_func: brun->info->bench_func_noreloc;
		bench(strms[i], dimGrid, dimBlock, (void **)((char *)d_args_brun + SIZE_ARGS * i), d_benches_res + i);
		err = hipGetLastError();
		if (err != hipSuccess) {
			printf("error: %s\n", hipGetErrorString(err));
			return FALSE;
		}
	}

	for (i = 0; i < n_benches; i++)
		hipStreamSynchronize(strms[i]);

	*pticks = get_tickcount();

	hipFree(d_args_brun);

	for (i = 0, brun = benchruns; i < n_benches; i++, brun++) {
		hipMemcpy(&brun->res, d_benches_res + i, sizeof(int), hipMemcpyDeviceToHost);
	}
	hipFree(d_benches_res);

	return TRUE;
}
