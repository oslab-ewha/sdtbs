#include "../benchapi.h"

extern __device__ int kmeans(void *args[]);

__device__ int
kmeans_reloc(void *args[])
{
	return kmeans(args);
}

int
bench_kmeans_reloc(hipStream_t strm, dim3 dimGrid, dim3 dimBlock, void *args[])
{
	void	**d_args;
	int	res, *d_pres;

	hipMalloc(&d_args, sizeof(void *) * 5);
	hipMalloc(&d_pres, sizeof(int));
	hipMemcpyAsync(d_args, args, sizeof(void *) * 5, hipMemcpyHostToDevice, strm);

	launch_kernel(KMEANS_RELOC, strm, dimGrid, dimBlock, d_args, d_pres);

	hipMemcpyAsync(&res, d_pres, sizeof(int), hipMemcpyDeviceToHost, strm);
	hipStreamSynchronize(strm);
	hipFree(d_args);
	hipFree(d_pres);

	return res;
}
