#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <unistd.h>

#include <hip/hip_math_constants.h>

#include "../../benchapi.h"

/* NOTE: nvcc over 9.0 has a problem of compilation freeze.
 * This seems to relate to ptx optimization.
 * nvcc with -Xptxas -O0 disables all optimizations. loopcalc Makefile.am has this option.
 */

__device__ static int
calc_empty(int n_iters)
{
        int     value = 0;
	int     i;

        for (i = 0; i < n_iters; i++) {
		value++;
        }
        return value;
}

__device__ static int
calc_int(int n_iters)
{
	int	value = 9;
	int     i;

        for (i = 0; i < n_iters; i++) {
		value = value * 1923192273 + 3921192123;
        }
        return value;
}

__device__ static float
calc_float(int n_iters)
{
	float	value = 9.99f;
	int     i;

        for (i = 0; i < n_iters; i++) {
		if (value == HIP_INF_F)
			value = 9.99f;
		else
			value = value * 2911.2123f + 1.992813f;
        }
        return value;
}

__device__ static double
calc_double(int n_iters)
{
	double	value = 32.192123123213;
	int	i;

	for (i = 0; i < n_iters; i++) {
		if (value == HIP_INF_F)
			value = 329.99128493;
		else
			value = value * 2911.2134324 + 1.992812932;
	}
	return value;
}

__device__ int
loopcalc(void *args[])
{
	int	calctype = (int)(long long)args[0];
	int	n_iters1 = (int)(long long)args[1];
	int	ret = 0;

	switch (calctype) {
	case 1:
		ret = (int)calc_int(n_iters1);
		break;
	case 2:
		ret = (int)fmodf(calc_float(n_iters1), 10000000.0f);
		break;
	case 3:
		ret = (int)fmod(calc_double(n_iters1), 100000000.0);
		break;
	default:
		ret = (int)calc_empty(n_iters1);
		break;
	}
	return ret;
}

int
bench_loopcalc(hipStream_t strm, dim3 dimGrid, dim3 dimBlock, void *args[])
{
	skrid_t	skrid;
	int	res;

	skrid = launch_kernel(LOOPCALC, strm, dimGrid, dimBlock, args);
	wait_kernel(skrid, strm, &res);

	return res;
}
