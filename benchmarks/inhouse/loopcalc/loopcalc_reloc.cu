#include "../../benchapi.h"

extern __device__ int loopcalc(void *args[]);

__device__ int
loopcalc_reloc(void *args[])
{
	return loopcalc(args);
}

int
bench_loopcalc_reloc(hipStream_t strm, dim3 dimGrid, dim3 dimBlock, void *args[])
{
	void    **d_args;
	int	res, *d_pres;

	hipMalloc(&d_args, sizeof(void *) * 2);
	hipMalloc(&d_pres, sizeof(int));
	hipMemcpyAsync(d_args, args, sizeof(void *) * 2, hipMemcpyHostToDevice, strm);

	launch_kernel(LOOPCALC_RELOC, strm, dimGrid, dimBlock, d_args, d_pres);

	hipMemcpyAsync(&res, d_pres, sizeof(int), hipMemcpyDeviceToHost, strm);
	hipStreamSynchronize(strm);
	hipFree(d_args);
	hipFree(d_pres);

	return res;
}
