#include "hip/hip_runtime.h"
#include "../../benchapi.h"

extern __device__ int gma(void *args[]);

__global__ static void
kernel_gma_reloc(void *args[], int *pres)
{
	*pres = gma(args);
}

void
bench_gma_reloc(hipStream_t strm, dim3 dimGrid, dim3 dimBlock, void *args[], int *pres)
{
	kernel_gma_reloc<<<dimGrid, dimBlock, 0, strm>>>(args, pres);
}
