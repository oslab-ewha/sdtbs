#include "../../benchapi.h"

extern __device__ int gma(void *args[]);

__device__ int
gma_reloc(void *args[])
{
	return gma(args);
}

int
bench_gma_reloc(hipStream_t strm, dim3 dimGrid, dim3 dimBlock, void *args[])
{
	void	**d_args;
	int	res, *d_pres;

	hipMalloc(&d_args, sizeof(void *) * 4);
	hipMalloc(&d_pres, sizeof(int));
	hipMemcpyAsync(d_args, args, sizeof(void *) * 3, hipMemcpyHostToDevice, strm);

	launch_kernel(GMA_RELOC, strm, dimGrid, dimBlock, d_args, d_pres);

	hipMemcpyAsync(&res, d_pres, sizeof(int), hipMemcpyDeviceToHost, strm);
	hipStreamSynchronize(strm);
	hipFree(d_args);
	hipFree(d_pres);

	return res;
}
