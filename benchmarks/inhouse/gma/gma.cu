
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>
#include <unistd.h>

__device__ static unsigned
rand_xorshift(unsigned seed)
{
	seed ^= (seed << 13);
	seed ^= (seed >> 17);
	seed ^= (seed << 5);
	return seed;
}

__device__ static unsigned
get_memidx(int memidx, int gmemsize)
{
	unsigned	value;

	value = (unsigned)(gmemsize * rand_xorshift(memidx * 19373193));
	return value % (gmemsize * 1024);
}

__device__ int
gma(void *args[])
{
	int	gmemsize = (int)(long long)args[0];
	int	n_iters = (int)(long long)args[1];
	unsigned char	*gmem = (unsigned char *)args[2];
	int	memidx;
	int	value = 0;
	int	i, j;

	memidx = get_memidx(391 + (threadIdx.x % 32) * 2913751, gmemsize);
	for (i = 0; i < n_iters; i++) {
		for (j = 0; j < 10000; j++) {
			memidx = get_memidx(memidx, gmemsize);
			value += gmem[memidx];
		}
	}
	return value;
}

__global__ static void
kernel_gma(void *args[])
{
	int	ret;

	ret = gma(args);
	if (threadIdx.x == 0 && threadIdx.y == 0)
		args[0] = (void *)(long long)ret;
}

int
cookarg_gma(void *args[])
{
	unsigned char	*gmem;
	int	gmemsize = (int)(long long)args[0];
	char	buf[1024];
	int	i;
	hipError_t	err;

	err = hipMalloc((void **)&gmem, gmemsize * 1024);
	if (err != hipSuccess) {
		printf("hipMalloc failed: err: %s\n", hipGetErrorString(err));
		return -1;
	}
	for (i = 0; i < 1024; i++) {
		buf[i] = i;
	}
	for (i = 0; i < gmemsize; i++) {
		hipMemcpy(gmem + i * 1024, buf, 1024, hipMemcpyHostToDevice);
	}
	args[2] = gmem;
	return 0;
}

int
bench_gma(hipStream_t strm, int n_grid_width, int n_grid_height, int n_tb_width, int n_tb_height, void *args[])
{
	hipError_t	err;

	dim3 dimGrid(n_grid_width, n_grid_height);
	dim3 dimBlock(n_tb_width, n_tb_height);

	kernel_gma<<<dimGrid, dimBlock, 0, strm>>>(args);

	err = hipGetLastError();
	if (err != hipSuccess) {
		printf("error: %s\n", hipGetErrorString(err));
		return -1;
	}

	return 0;
}
