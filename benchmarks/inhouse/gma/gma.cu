#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <unistd.h>

#include "../../benchapi.h"

__device__ int
gma(void *args[])
{
	int	gmemsize = (int)(long long)args[0];
	int	stride = (int)(long long)args[1];
	int	n_iters = (int)(long long)args[2];
	unsigned char	*gmem = (unsigned char *)args[3];
	unsigned	memidx_max = gmemsize * 1024;
	unsigned	memidx;
	int	value = 0;
	int	i;

	memidx = (unsigned)(clock() * 19239913 * get_threadIdxX()) % memidx_max;
	for (i = 0; i < n_iters; i++, memidx += stride) {
		if (memidx >= memidx_max)
			memidx -= memidx_max;
		value += (gmem[memidx] + gmem[memidx + stride / 2] + gmem[memidx + stride / 4]);
	}
	return value;
}

int
cookarg_gma(dim3 dimGrid, dim3 dimBlock, void *args[])
{
	unsigned char	*gmem;
	int	gmemsize = (int)(long long)args[0];
	char	buf[1024];
	int	i;
	hipError_t	err;

	err = hipMalloc((void **)&gmem, gmemsize * 1024);
	if (err != hipSuccess) {
		printf("hipMalloc failed: err: %s\n", hipGetErrorString(err));
		return -1;
	}
	for (i = 0; i < 1024; i++) {
		buf[i] = i;
	}
	for (i = 0; i < gmemsize; i++) {
		hipMemcpy(gmem + i * 1024, buf, 1024, hipMemcpyHostToDevice);
	}
	args[3] = gmem;
	return 0;
}

__global__ static void
kernel_gma(void *args[], int *pres)
{
	native_mode = 1;
	*pres = gma(args);
}

void
bench_gma(hipStream_t strm, dim3 dimGrid, dim3 dimBlock, void *args[], int *pres)
{
	kernel_gma<<<dimGrid, dimBlock, 0, strm>>>(args, pres);
}
