#include "../../benchapi.h"

extern __device__ int lma(void *args[]);

__device__ int
lma_reloc(void *args[])
{
	return lma(args);
}

int
bench_lma_reloc(hipStream_t strm, dim3 dimGrid, dim3 dimBlock, void *args[])
{
	void    **d_args;
	int	res, *d_pres;

	hipMalloc(&d_args, sizeof(void *) * 4);
	hipMalloc(&d_pres, sizeof(int));
	hipMemcpyAsync(d_args, args, sizeof(void *) * 4, hipMemcpyHostToDevice, strm);

	launch_kernel(LMA_RELOC, strm, dimGrid, dimBlock, d_args, d_pres);

	hipMemcpyAsync(&res, d_pres, sizeof(int), hipMemcpyDeviceToHost, strm);
	hipStreamSynchronize(strm);
	hipFree(d_args);
	hipFree(d_pres);

	return res;
}
