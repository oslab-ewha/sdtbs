#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <unistd.h>

#include "../../benchapi.h"

__device__ int
lma(void *args[])
{
	int	chunksize = (int)(long long)args[0];
	int	refspan = (int)(long long)args[1];
	int	n_iters = (int)(long long)args[2];
	int	chunk_idx = get_blockIdxX() + get_gridDimX() * get_blockIdxY();
	int	chunk_start;
	unsigned char	**chunks = (unsigned char **)args[3];
	unsigned	n_chunks = get_gridDimX() * get_gridDimY();
	unsigned	randx;
	int	value = 0;
	int	i;

	chunk_start = chunk_idx - refspan;
	if (chunk_start < 0)
		chunk_start += n_chunks;

	randx = 0x12345678 + clock() * 19239913 * get_threadIdxX();
	for (i = 0; i < n_iters; i++) {
		unsigned	rand_chunk_idx = (chunk_start + randx % (1 + 2 * refspan)) % n_chunks;

		randx = get_random(randx);
		value += chunks[rand_chunk_idx][randx % chunksize];
		randx = get_random(randx);
	}
	return value;
}

int
cookarg_lma(dim3 dimGrid, dim3 dimBlock, void *args[])
{
	unsigned char	**chunks, **d_chunks;
	int	chunksize = (int)(long long)args[0];
	int	refspan = (int)(long long)args[1];
	char	*buf;
	int	i;
	hipError_t	err;

	if (dimGrid.x * dimGrid.y < refspan * 2) {
		printf("too small TB's\n");
		return -1;
	}
	chunks = (unsigned char **)malloc(dimGrid.x * dimGrid.y * sizeof(unsigned char *));
	buf = (char *)malloc(chunksize);
	for (i = 0; i < chunksize; i++) {
		buf[i] = (char)i;
	}
	for (i = 0; i < dimGrid.x * dimGrid.y; i++) {
		err = hipMalloc(&chunks[i], chunksize);
		if (err != hipSuccess) {
			printf("hipMalloc failed: err: %s\n", hipGetErrorString(err));
		}
		hipMemcpy(chunks[i], buf, chunksize, hipMemcpyHostToDevice);
	}
	free(buf);

	hipMalloc(&d_chunks, dimGrid.x * dimGrid.y * sizeof(unsigned char *));
	hipMemcpy(d_chunks, chunks, dimGrid.x * dimGrid.y * sizeof(unsigned char *), hipMemcpyHostToDevice);

	args[3] = d_chunks;
	return 0;
}

int
bench_lma(hipStream_t strm, dim3 dimGrid, dim3 dimBlock, void *args[])
{
	skrid_t	skrid;
	int	res;

	skrid = launch_kernel(LMA, strm, dimGrid, dimBlock, args);
	wait_kernel(skrid, strm, &res);

	return res;
}
